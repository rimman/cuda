#include "hip/hip_runtime.h"
#include "kernel.h"
#define TPB 64
#define RAD 1 // radius of the stencil

///////////////////////////////////////////////////////
// Note: Setting size of the Shared Array
// if you createyour shared array with a fixed size, the array can be created as follows:

// __shared__ float s_in[34];
//
// and not change to the kernel call is needed. Note that teh following will produce a compiler error:
//
// __shared__ float s_in[blockDim.x + 2 * RAD];
//
// if you allocate the array dynamically, the declaration requires the keyword "extern" as follows:
//
// extern __shared__ float s_in[];
//
// And the kernel call requires an optional third argument within the chevrons to specify the size
// of the shared memory allocation in bytes.
// 
// const size_t smemSize = (TPB + 2 * RAD) * sizeof(float);
// ddKernel<<<n+TPB-1)/TPB, TPB, smemSize>>>(d_out, d_in, n, h);

__global__
void ddKernel(float *d_out, const float *d_in, int size, float h) 
{
    // compute "global index" i
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
  
    if (i >= size) return;
    
    const int s_idx = threadIdx.x + RAD;
    
    //declare shared array
    extern __shared__ float s_in[];

    // Regular cells 
    // each thread requests the entry in the input array whose index matches 
    // the thread's global index and stores the value int eh shared array at the entry corresponding to the local index)
    s_in[s_idx] = d_in[i];

    // Halo cells
    if (threadIdx.x < RAD) 
    {
        // careful: the two lines below will also access d_in[-1] and d_in[size+1] which 
        // are undefined! This bug is fixed in heat_2d (cf. idxClip function)
    
        s_in[s_idx - RAD] = d_in[i - RAD];
        s_in[s_idx + blockDim.x] = d_in[i + blockDim.x];
    }
  
    //Kernel launches are async. We cannot assume that all of the input data
    //has been loaded into the shared memory array before threads execute the
    //final statement. To ensure that all the data has been properly stored,
    //call __syncthreads(), which forces all the threads in the block to complete
    //the previous statements ebfore any thread in the block poceeds further.
    __syncthreads();
    d_out[i] = (s_in[s_idx-1] - 2.f*s_in[s_idx] + s_in[s_idx+1])/(h*h);
}

void ddParallel(float *out, const float *in, int n, float h) 
{
  float *d_in = 0, *d_out = 0;
  hipMalloc(&d_in, n * sizeof(float));
  hipMalloc(&d_out, n * sizeof(float));
  hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);

  // Set shared memory size in bytes
  const size_t smemSize = (TPB + 2 * RAD) * sizeof(float);

  ddKernel<<<(n + TPB - 1)/TPB, TPB, smemSize>>>(d_out, d_in, n, h);

  hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}